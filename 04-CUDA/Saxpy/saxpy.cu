#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"

#include "omp.h"

#include <chrono>

using namespace std;

__global__ void gpu_saxpy(int n, float a, float *x, float *y, float *s)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) s[i] = a*x[i] + y[i];
}

void cpu_saxpy(int n, float a, float *x, float *y, float *s)
{
  #pragma omp parallel for num_threads(12) 
  for (int i=0; i<n; i++)
  {
      s[i] = a*x[i] + y[i];
  }
}

void cpu_saxpy_mono(int n, float a, float *x, float *y, float *s)
{
  for (int i=0; i<n; i++)
  {
      s[i] = a*x[i] + y[i];
  }
}


int main(void)
{
  unsigned long int N = 4096*4096*16;
  float *x, *y, *s_cpu, *s_gpu, *d_x, *d_y, *d_s;
  
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  s_gpu = (float*)malloc(N*sizeof(float));
  s_cpu = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_s, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  std::chrono::high_resolution_clock::time_point t0 = std::chrono::high_resolution_clock::now();
  cpu_saxpy(N, 2.0f, x, y, s_cpu);
  std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
  auto cpu_duration = std::chrono::duration<double>(t1-t0).count();

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  int k = 32; 
  t0 = std::chrono::high_resolution_clock::now();
  gpu_saxpy<<<(N+k)/k, k>>>(N, 2.0f, d_x, d_y, d_s);  
  t1 = std::chrono::high_resolution_clock::now();
  auto gpu_duration = std::chrono::duration<double>(t1-t0).count();
  hipMemcpy(s_gpu, d_s, N*sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = std::max(maxError, s_cpu[i]-s_gpu[i]);
  printf("Max error: %f\n", maxError);

  printf("cpu_duration: %f\n", cpu_duration);
  printf("gpu_duration: %f\n", gpu_duration);


  k = 512; 
  t0 = std::chrono::high_resolution_clock::now();
  gpu_saxpy<<<(N+k)/k, k>>>(N, 2.0f, d_x, d_y, d_s);  
  t1 = std::chrono::high_resolution_clock::now();
  gpu_duration = std::chrono::duration<double>(t1-t0).count();
  hipMemcpy(s_gpu, d_s, N*sizeof(float), hipMemcpyDeviceToHost);

  maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = std::max(maxError, s_cpu[i]-s_gpu[i]);
  printf("Max error: %f\n", maxError);

  printf("cpu_duration: %f\n", cpu_duration);
  printf("gpu_duration: %f\n", gpu_duration);


  k = 2048; 
  t0 = std::chrono::high_resolution_clock::now();
  gpu_saxpy<<<(N+k)/k, k>>>(N, 2.0f, d_x, d_y, d_s);  
  t1 = std::chrono::high_resolution_clock::now();
  gpu_duration = std::chrono::duration<double>(t1-t0).count();
  hipMemcpy(s_gpu, d_s, N*sizeof(float), hipMemcpyDeviceToHost);
  
  maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = std::max(maxError, s_cpu[i]-s_gpu[i]);
  printf("Max error: %f\n", maxError);

  printf("cpu_duration: %f\n", cpu_duration);
  printf("gpu_duration: %f\n", gpu_duration);
  
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_s);
  free(x);
  free(y);
  free(s_cpu);
  free(s_gpu);
}

